#include "hip/hip_runtime.h"
#include "cclcuda.cuh"
#include <cmath>
#include <>
#include <iostream>
#include <iomanip>

const int BLOCK = 8;

__device__ int IMin(int a, int b)
{
    return a < b ? a : b;
}

__global__ void InitCCL(int labelList[], int reference[], int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int id = x + y * width;

    labelList[id] = reference[id] = id;
}

__global__ void scanning8(unsigned char frame[], int labelList[], int reference[], bool* markFlag, int N, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int id = x + y * width;

    if (id >= N)
        return;

    unsigned char value = frame[id];
    int label = N;

    if (id - width >= 0 && value == frame[id - width])
        label = IMin(label, labelList[id - width]);
    if (id + width < N  && value == frame[id + width])
        label = IMin(label, labelList[id + width]);

    int col = id % width;
    if (col > 0)
    {
        if (value == frame[id - 1])
            label = IMin(label, labelList[id - 1]);
        if (id - width - 1 >= 0 && value == frame[id - width - 1])
            label = IMin(label, labelList[id - width - 1]);
        if (id + width - 1 < N  && value == frame[id + width - 1])
            label = IMin(label, labelList[id + width - 1]);
    }
    if (col + 1 < width)
    {
        if (value == frame[id + 1])
            label = IMin(label, labelList[id + 1]);
        if (id - width + 1 >= 0 && value == frame[id - width + 1])
            label = IMin(label, labelList[id - width + 1]);
        if (id + width + 1 < N  && value == frame[id + width + 1])
            label = IMin(label, labelList[id + width + 1]);
    }

    if (label < labelList[id])
    {
        reference[labelList[id]] = label;
        *markFlag = true;
    }
}

__global__ void analysis(int labelList[], int reference[], int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int id = x + y * width;

    int label = labelList[id];
    int ref;
    if (label == id)
    {
        do
        {
            ref = label;
            label = reference[ref];
        } while (ref ^ label);
        reference[id] = label;
    }
}

__global__ void labeling(int labelList[], int reference[], int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int id = x + y * width;

    labelList[id] = reference[reference[labelList[id]]];
}

void CCLLEGPU::CudaCCL(unsigned char* frame, int* labels, int width, int height, int degreeOfConnectivity, unsigned char threshold)
{
    auto N = width * height;

    hipMalloc(reinterpret_cast<void**>(&LabelListOnDevice), sizeof(int) * N);
    hipMalloc(reinterpret_cast<void**>(&ReferenceOnDevice), sizeof(int) * N);
    hipMalloc(reinterpret_cast<void**>(&FrameDataOnDevice), sizeof(unsigned char) * N);

    hipMemcpy(FrameDataOnDevice, frame, sizeof(unsigned char) * N, hipMemcpyHostToDevice);

    bool* markFlagOnDevice;
    hipMalloc(reinterpret_cast<void**>(&markFlagOnDevice), sizeof(bool));

    dim3 grid((width + BLOCK - 1) / BLOCK, (height + BLOCK - 1) / BLOCK);
    dim3 threads(BLOCK, BLOCK);

    InitCCL <<<grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);

    while (true)
    {
        auto markFalgOnHost = false;
        hipMemcpy(markFlagOnDevice, &markFalgOnHost, sizeof(bool), hipMemcpyHostToDevice);

        scanning8 <<< grid, threads >>>(FrameDataOnDevice, LabelListOnDevice, ReferenceOnDevice, markFlagOnDevice, N, width, height);

        hipDeviceSynchronize();
        hipMemcpy(&markFalgOnHost, markFlagOnDevice, sizeof(bool), hipMemcpyDeviceToHost);

        if (markFalgOnHost)
        {
            analysis <<< grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);
            hipDeviceSynchronize();
            labeling <<< grid, threads >>>(LabelListOnDevice, ReferenceOnDevice, width, height);
        }
        else
        {
            break;
        }
    }

    hipMemcpy(labels, LabelListOnDevice, sizeof(int) * N, hipMemcpyDeviceToHost);

    hipFree(FrameDataOnDevice);
    hipFree(LabelListOnDevice);
    hipFree(ReferenceOnDevice);
}
